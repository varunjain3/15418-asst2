#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#include "exclusiveScan.cu_inl"
// #include "circleBoxTest.cu_inl"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

#define TILE_SIZE 32
#define BLOCKSIZE TILE_SIZE *TILE_SIZE
#define NUM_CIRCLES BLOCKSIZE

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

// This stores the global constants
struct GlobalConstants
{

    SceneName sceneName;

    int numberOfCircles;

    float *position;
    float *velocity;
    float *color;
    float *radius;

    int imageWidth;
    int imageHeight;
    float *imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// Read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// Color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// Include parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake()
{

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a)
{

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update positions of fireworks
__global__ void kernelAdvanceFireWorks()
{
    const float dt = 1.f / 60.f;
    const float pi = M_PI;
    const float maxDist = 0.25f;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;
    float *radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS)
    { // firework center; no update
        return;
    }

    // Determine the firework center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i + 1];

    // Update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j + 1] += velocity[index3j + 1] * dt;

    // Firework sparks
    float sx = position[index3j];
    float sy = position[index3j + 1];

    // Compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // Compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist)
    { // restore to starting position
        // Random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j + 1] = position[index3i + 1] + y;
        position[index3j + 2] = 0.0f;

        // Travel scaled unit length
        velocity[index3j] = cosA / 5.0;
        velocity[index3j + 1] = sinA / 5.0;
        velocity[index3j + 2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // Place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff)
    {
        radius[index] = 0.02f;
    }
    else
    {
        radius[index] += 0.01f;
    }
}

// kernelAdvanceBouncingBalls
//
// Update the position of the balls
__global__ void kernelAdvanceBouncingBalls()
{
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3 + 1];
    float oldPosition = position[index3 + 1];

    if (oldVelocity == 0.f && oldPosition == 0.f)
    { // stop-condition
        return;
    }

    if (position[index3 + 1] < 0 && oldVelocity < 0.f)
    { // bounce ball
        velocity[index3 + 1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3 + 1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3 + 1] += velocity[index3 + 1] * dt;

    if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon && oldPosition < 0.0f && fabsf(position[index3 + 1] - oldPosition) < epsilon)
    { // stop ball
        velocity[index3 + 1] = 0.f;
        position[index3 + 1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// Move the snowflake animation forward one time step.  Update circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake()
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float *positionPtr = &cuConstRendererParams.position[index3];
    float *velocityPtr = &cuConstRendererParams.velocity[index3];

    // Load from global memory
    float3 position = *((float3 *)positionPtr);
    float3 velocity = *((float3 *)velocityPtr);

    // Hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // Add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // Drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // Update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // Update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // If the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ((position.y + radius < 0.f) ||
        (position.x + radius) < -0.f ||
        (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // Restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // Store updated positions and velocities to global memory
    *((float3 *)positionPtr) = position;
    *((float3 *)velocityPtr) = velocity;
}

__device__ __inline__ void
eachCircle(int n, short pixelX, short pixelY, float invWidth, float invHeight, float4 &rgb)
{   
    if (n >= cuConstRendererParams.numberOfCircles)
        return;
    float3 p = *(float3 *)(&cuConstRendererParams.position[3 * n]);
    float rad = cuConstRendererParams.radius[n];

    float diffX = p.x - invWidth * (static_cast<float>(pixelX) + 0.5f);
    float diffY = p.y - invHeight * (static_cast<float>(pixelY) + 0.5f);
    float pixelDist = diffX * diffX + diffY * diffY;

    if (pixelDist <= rad * rad)
    {
        float3 tempRGB = *(float3 *)&(cuConstRendererParams.color[3 * n]);
        float alpha = .5f;
        rgb.x = alpha * tempRGB.x + (1.f - alpha) * rgb.x;
        rgb.y = alpha * tempRGB.y + (1.f - alpha) * rgb.y;
        rgb.z = alpha * tempRGB.z + (1.f - alpha) * rgb.z;
        rgb.w = alpha + rgb.w;
    }
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles()
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // // Compute the bounding box of the circle. The bound is in integer
    // // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    int numCircles = cuConstRendererParams.numberOfCircles;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    __shared__ uint prefixSumInput[BLOCKSIZE];
    __shared__ uint prefixSumOutput[BLOCKSIZE + 1];
    __shared__ uint prefixSumScratch[2 * BLOCKSIZE];

    // if (index < cuConstRendererParams.numberOfCircles)
    // {
    //     p[threadIdx.x] = *(float3 *)(&cuConstRendererParams.position[3 * index]);
    //     rad[threadIdx.x] = cuConstRendererParams.radius[index];
    //     // tempRGB[threadIdx.x] = *(float3 *)&(cuConstRendererParams.color[3 * index]);
    // }
    // __syncthreads();

    // a local storage for each thread to store the rgb value
    uint size = imageWidth * imageHeight / BLOCKSIZE + 1;
    float4 sharedRGB[BLOCKSIZE];

    for (uint tileX = 0; tileX < imageWidth; tileX += 32)
    {
        for (uint tileY = 0; tileY < imageHeight; tileY += 32)
        {
            short pixelX = tileX + threadIdx.x % 32;
            short pixelY = tileY + threadIdx.x / 32;

            float4 rgb = make_float4(1.f, 1.f, 1.f, 1.f);
            int flagEdit = false;

            // if (circleInBox(
            //     float circleX, float circleY, float circleRadius,
            //     tileX, tileX+32, tileY, tileY+32))

            eachCircle(0, pixelX, pixelY, invWidth, invHeight, rgb);
            eachCircle(1, pixelX, pixelY, invWidth, invHeight, rgb);
            eachCircle(2, pixelX, pixelY, invWidth, invHeight, rgb);
            eachCircle(3, pixelX, pixelY, invWidth, invHeight, rgb);

            // printf("rgb: %f %f %f %f\n", rgb.x, rgb.y, rgb.z, rgb.w);
            int offset = 4 * (pixelY * imageWidth + pixelX);
            *(float4 *)(&cuConstRendererParams.imageData[offset]) = rgb;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer()
{
    image = NULL;

    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer()
{

    if (image)
    {
        delete image;
    }

    if (position)
    {
        delete[] position;
        delete[] velocity;
        delete[] color;
        delete[] radius;
    }

    if (cudaDevicePosition)
    {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image *
CudaRenderer::getImage()
{

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadScene(SceneName scene)
{
    sceneName = scene;
    loadCircleScene(sceneName, numberOfCircles, position, velocity, color, radius);
}

void CudaRenderer::setup()
{

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numberOfCircles = numberOfCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // Also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int *permX;
    int *permY;
    float *value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // Copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height)
{

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage()
{

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME)
    {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    }
    else
    {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES)
    {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    }
    else if (sceneName == BOUNCING_BALLS)
    {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    }
    else if (sceneName == HYPNOSIS)
    {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    }
    else if (sceneName == FIREWORKS)
    {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void CudaRenderer::render()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(1024, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    kernelRenderCircles<<<gridDim, blockDim>>>();
    cudaCheckError(hipDeviceSynchronize());
}